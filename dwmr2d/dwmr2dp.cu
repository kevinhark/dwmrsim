
#include <hip/hip_runtime.h>
__global__ void ftcsKernel(float *Cxn, float *Cyn, float *Cxo, float *Cyo, float *diffu,float *diffd,float *diffl, float *diffr, float *T2val, float Adx, int dimX)
{
	int x = threadIdx.x + blockDim.x*blockIdx.x;// place in x dim
	int y = blockIdx.y; 						// place in y dim
	int ind = x+y*dimX;							// current index in linear space
	int yp1 = y+1;
	int ym1 = y-1;
	int xp1 = x+1;
	int xm1 = x-1;
	
	// periodic bc
	if (y==0) ym1 = gridDim.y-1;
	if (y==gridDim.y-1) yp1 = 0;
	if (x==0) xm1 = dimX-1;
	if (x==dimX-1) xp1 = 0;

	if (x >= 0 && x <= (dimX-1) && y >= 0 && y <= (gridDim.y-1) )
	{
		Cxn[ind] = Cxo[ind] - T2val[ind]*Cxo[ind]
			+ diffu[ind]*(cos(Adx)*Cxo[yp1*dimX+x] + sin(Adx)*Cyo[yp1*dimX+x] - Cxo[ind])
			+ diffd[ind]*(cos(Adx)*Cxo[ym1*dimX+x] - sin(Adx)*Cyo[ym1*dimX+x] - Cxo[ind])
			+ diffl[ind]*(Cxo[y*dimX+xp1] - Cxo[ind])
			+ diffr[ind]*(Cxo[y*dimX+xm1] - Cxo[ind]);
			
		Cyn[ind] = Cyo[ind] - T2val[ind]*Cyo[ind]
			+ diffu[ind]*(cos(Adx)*Cyo[yp1*dimX+x] - sin(Adx)*Cxo[yp1*dimX+x] - Cyo[ind])
			+ diffd[ind]*(cos(Adx)*Cyo[ym1*dimX+x] + sin(Adx)*Cxo[ym1*dimX+x] - Cyo[ind])
			+ diffl[ind]*(Cyo[y*dimX+xp1] - Cyo[ind])
			+ diffr[ind]*(Cyo[y*dimX+xm1] - Cyo[ind]);
	}
}
